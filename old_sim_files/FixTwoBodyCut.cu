#include "hip/hip_runtime.h"
#include "FixTwoBodyCut.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "State.h"
#include "cutils_func.h"
#include "ReadConfig.h"
#include "EvaluatorWrapper.h"
#include "PairEvaluatorTwoBody.h"
#include "EvaluatorWrapper.h"
//#include "ChargeEvaluatorEwald.h"
using namespace std;
namespace py = boost::python;
const string TwoBodyCutType = "TwoBodyCut";



FixTwoBodyCut::FixTwoBodyCut(boost::shared_ptr<State> state_, string handle_)
    : FixPair(state_, handle_, "all", TwoBodyCutType, true, false, 1),
    epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut"), sigma_RHandle("sigma_R"), r_AHandle("r_A"), alpha_AHandle("alpha_A"), GHandle("G"), r_GHandle(r_G), sigma_GHandle("sigma_G")
{

    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    initializeParameters(sigma_RHandle, sigma_Rs);
    initializeParameters(r_AHandle, r_As);
    initializeParameters(alpha_AHandle, alpha_As);
    initializeParameters(GHandle, Gs);
    initializeParameters(r_GHandle, r_Gs);
    initializeParameters(sigma_GHandle, sigma_Gs);
    paramOrder = {rCutHandle, epsHandle, sigHandle, sigma_RHandle, r_AHandle, alpha_AHandle, GHandle, r_GHandle, sigma_GHandle};
    readFromRestart();
    canAcceptChargePairCalc = true;
    setEvalWrapper();
}

void FixTwoBodyCut::compute(int virialMode) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

    evalWrap->compute(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx),
                      neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
                      state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
                      neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.virials.d_data.data(), gpd.qs(activeIdx), chargeRCut, virialMode);

}

void FixTwoBodyCut::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;
    evalWrap->energy(nAtoms, gpd.xs(activeIdx), perParticleEng, neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], gpd.qs(activeIdx), chargeRCut);

}

void FixTwoBodyCut::setEvalWrapper() {
    if (evalWrapperMode == "offload") {
        EvaluatorTwoBody eval;
        evalWrap = pickEvaluator<EvaluatorTwoBody, 8, true>(eval, chargeCalcFix);
    } else if (evalWrapperMode == "self") {
        EvaluatorTwoBody eval;
        evalWrap = pickEvaluator<EvaluatorTwoBody, 8, true>(eval, nullptr);
    }
}

bool FixTwoBodyCut::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillEps = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillSig = [] (float a, float b) {
            return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    auto processSigma_R = [] (float a) {
        return a;
    };
    auto processR_A = [] (float a) {
        return a;
    };
    auto processAlpha_A = [] (float a) {
        return 1.0f / a;
    };
    auto processG = [] (float a) {
        return a;
    };
    auto processR_G = [] (float a) {
        return a;
    };
    auto processSigma_G = [] (float a) {
        return 2.0f / (a*a);
    };
    prepareParameters(epsHandle, fillEps, processEps, false);
    prepareParameters(sigHandle, fillSig, processSig, false);
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    prepareParameters(sigma_RHandle, fillEps, processSigma_R, false);
    prepareParameters(r_AHandle, fillEps, processR_A, false);
    prepareParameters(alpha_AHandle, fillEps, processAlpha_A, false);
    prepareParameters(GHandle, fillEps, processG, false);
    prepareParameters(r_GHandle, fillEps, processR_G, false);
    prepareParameters(sigma_GHandle, fillEps, processSigma_G, false);

    sendAllToDevice();
    setEvalWrapper();
    return true;
}

string FixTwoBodyCut::restartChunk(string format) {
    stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}


bool FixTwoBodyCut::postRun() {

    return true;
}

void FixTwoBodyCut::addSpecies(string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    initializeParameters(sigma_RHandle, sigma_Rs);
    initializeParameters(r_AHandle, r_As);
    initializeParameters(alpha_AHandle, alpha_As);
    initializeParameters(GHandle, Gs);
    initializeParameters(r_GHandle, r_Gs);
    initializeParameters(sigma_GHandle, sigma_Gs);

}

vector<float> FixTwoBodyCut::getRCuts() {
    vector<float> res;
    vector<float> &src = *(paramMap[rCutHandle]);
    for (float x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

void export_FixTwoBodyCut() {
    py::class_<FixTwoBodyCut, boost::shared_ptr<FixTwoBodyCut>, py::bases<FixPair>, boost::noncopyable > (
        "FixTwoBodyCut",
        py::init<boost::shared_ptr<State>, string> (py::args("state", "handle"))
    )
      ;
}
